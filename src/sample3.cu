#include "hip/hip_runtime.h"
//*****************************************************************************
// sample3.cu
// 
// Expected output of the TXL transformation from sample3.c
//
// Project Description: A TXL transformation from OpenMP C sources to CUDA 
// equivalent. 
//
// For more information on TXL, visit: txl.ca
// Authors: AmirHossein Sojoodi, Nicolas Merz
// Course: ELEC-875 2020, Tom Dean
// Queen's University
//*****************************************************************************

#include<stdlib.h>
#include<omp.h>
#include<stdio.h>

#define SIZE 10000
#define PI 3.14
#define BLOCK_SIZE 64

__global__ void kernel(int *array1, int *array2, float *array3, int s) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < s) {
        array1[i] = i;
        array2[i] = array3[i] + array1[i];
        array2[i] += 10;
		array4[i] = foo * array2[i];
    }
}


int main(){

	int *array1;
    int *array2;
    float *array3;
    float *array4;
    int size = SIZE;
	
    hipMallocManaged(&array1, size * sizeof (int));
    hipMallocManaged(&array2, size * sizeof (int));
    hipMallocManaged(&array3, size * sizeof (int));
    array4 = (float *) malloc (size * sizeof (float));
	
    kernel<<<((size) - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(array1, array2, array3, size);
    hipDeviceSynchronize();
	
    printf ("Array3[%d] = %f", size - 1, array3[size - 1]);
    
	hipFree(array1);
    hipFree(array2);
    hipFree(array3);
    free(array4);
    
	return 0;
}

