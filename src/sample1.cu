//*****************************************************************************
// sample1.cu
// 
// Expected output of the TXL transformation from sample1.c
//
// Project Description: A TXL transformation from OpenMP C sources to CUDA 
// equivalent. 
//
// For more information on TXL, visit: txl.ca
// Authors: AmirHossein Sojoodi, Nicolas Merz
// Course: ELEC-875 2020, Tom Dean
// Queen's University
//*****************************************************************************


#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<omp.h>
#include<stdio.h>

#define SIZE 10000
#define BLOCK_SIZE 64

__global__ void kernel(int *array, int s){
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < s) {
		array[i] = i;
	}
}

int main(){

	int *array;
	int size = SIZE;

	hipMallocManaged(&array, size * sizeof(int));
	
	// Compute ceil(size/blockSize)
	kernel<<<(size - 1)/BLOCK_SIZE + 1, BLOCK_SIZE>>>(array, size);	
	hipDeviceSynchronize();
	
	// A simple validity test
	printf("Array[%d] = %d", size - 1, array[size - 1]);

	hipFree(array);

	return 0;
}

